#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "add_vector.hpp"

using std::cout;
using std::endl;

__global__ void add_vector(float* out, float* a, float* b, int n)
{
  // int i = blockIdx.x * blockDim.x + threadIdx.x;
  // for (int i = 0; i < n; i++)
  // {
  //   out[i] = a[i] + b[i];
  // }

  int index = 0;
  int stride = blockDim.x;

  // index = threadIdx.x + blockIdx.x * blockDim.x;
  // index = threadIdx.x + threadIdx.x * blockDim.x;

  for (int i = threadIdx.x; i < n; i += stride)
  {
    // out[i] = a[i] + b[i];
    out[i] = threadIdx.x;
  }

  // out[threadIdx.x] = threadIdx.x;
}

int cu_add_vector(float* out, float* a, float* b, int N)
{
  float *d_a, *d_b, *d_out;
  hipError_t err;

  hipMalloc(&d_out, N * sizeof(float));
  hipMalloc(&d_a, N * sizeof(float));
  hipMalloc(&d_b, N * sizeof(float));

  hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, N * sizeof(float), hipMemcpyHostToDevice);

  // block, threads
  // add_vector<<<1, 1>>>(d_out, d_a, d_b, N);
  add_vector<<<1, 8>>>(d_out, d_a, d_b, N);

  hipError_t errSync = hipGetLastError();
  hipError_t errAsync = hipDeviceSynchronize();
  if (errSync != hipSuccess)
  {
    printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
  }
  if (errAsync != hipSuccess)
  {
    printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
  }

  hipDeviceSynchronize();

  err = hipMemcpy(out, d_out, N * sizeof(float), hipMemcpyDeviceToHost);
  if (err != hipSuccess)
  {
    cout << "err: " << hipGetErrorString(err) << endl;
  }

  hipFree(d_out);
  hipFree(d_a);
  hipFree(d_b);

  return 0;
}